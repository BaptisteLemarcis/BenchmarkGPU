#include "hip/hip_runtime.h"
#include <tuple>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <>

#include "Logger.h"
#include "SoftmaxLayer.h"
#include "GenericFunctions.h"

__global__ void SoftmaxLossKernel(float* d_target, float* d_loss_data, int size) {
	//int idxTarget = (blockDim.x * blockIdx.x + threadIdx.x) + blockIdx.y;
	int idx = ((blockDim.x * blockIdx.x + threadIdx.x) * blockDim.y) + blockIdx.y;

	if (idx >= size) return;

	if (d_target[idx] == 1.f)
		d_loss_data[idx] -= 1.f;
}

__global__ void SoftmaxErrorKernel(float* d_error, float* d_input, float* d_targets, int size) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idx1 = 2 * idx;
	int idx2 = (2 * (idx+1))-1;

	/*int idx3 = 2 * (idx+1);
	int idx4 = (2 * ((idx+1) + 1)) - 1;*/

	if (idx <= size) return;
	float z = d_input[idx1] - d_targets[idx1];
	*d_error += z*z;
	z = d_input[idx2] - d_targets[idx2];
	*d_error += z*z;
}

SoftmaxLayer::SoftmaxLayer(int inputDim, int batchSize) : Layer(inputDim, inputDim, batchSize)
{
	//
	// Create Descriptor
	//
	CheckError(hipdnnCreateTensorDescriptor(&m_outputDesc), __FILE__, __LINE__);

	// Allocating
	CheckError(hipMalloc(&m_d_output, m_outputDim*batchSize * sizeof(float)), __FILE__, __LINE__);

	//
	// Setting up TensorDescriptor
	//
	int dimA[3];
	int strideA[3];
	dimA[0] = batchSize;
	dimA[1] = m_outputDim;
	dimA[2] = 1;

	strideA[0] = dimA[2] * dimA[1];
	strideA[1] = dimA[2];
	strideA[2] = 1;

	CheckError(hipdnnSetTensorNdDescriptor(m_outputDesc, HIPDNN_DATA_FLOAT, 3, dimA, strideA), __FILE__, __LINE__);

	CheckError(hipDeviceSynchronize(), __FILE__, __LINE__);

	/*delete[] dimA;
	delete[] strideA;*/
}

SoftmaxLayer::~SoftmaxLayer()
{
	CheckError(hipFree(&m_d_output), __FILE__, __LINE__);
	CheckError(hipdnnDestroyTensorDescriptor(m_outputDesc), __FILE__, __LINE__);
}

std::tuple<float, float*> SoftmaxLayer::forward(hipdnnHandle_t& handle, hipblasHandle_t& cublasHandle, float* d_input, float* d_targets, float* d_onevec)
{
	float alpha = float(1);
	float beta = float(0);

	CheckError(hipdnnSoftmaxForward(handle,
		HIPDNN_SOFTMAX_ACCURATE,
		HIPDNN_SOFTMAX_MODE_INSTANCE,
		&alpha,
		m_outputDesc, // m_dstDataDesc[0]
		d_input,
		&beta,
		m_outputDesc,
		m_d_output), __FILE__, __LINE__);

	//printDeviceVectorToFile(2, m_output, 0);
	float* d_batchError, batchError;
	CheckError(hipMalloc(&d_batchError, sizeof(float)), __FILE__, __LINE__);

	SoftmaxErrorKernel <<<RoundUp(m_batchSize, 128), 128 >>> (d_batchError, d_input, d_targets, m_batchSize);

	CheckError(hipMemcpy(&batchError, d_batchError, sizeof(float), hipMemcpyDeviceToHost), __FILE__, __LINE__);
	
	/*for (int i = 0; i < m_batchSize; i++) {
		float* output = new float[getOutputDim() * m_batchSize];
		float* target = targets[i];
		CheckError(hipMemcpy(output, input, getOutputDim() * m_batchSize * sizeof(float), hipMemcpyDeviceToHost), __FILE__, __LINE__);
		float sum = 0.f;
		for (int i = 0; i < getOutputDim(); i++) {
			float z = output[i] - target[i];
			sum += z*z;
		}
		sum /= getOutputDim();
		batchError += sum;
	}*/
	batchError /= m_batchSize;

	/*if (FLAGS_DEBUG) {
		logFile << "\tResulting softmax data" << std::endl;
		printDeviceVectorToFile(NBDATADSP, (float *)m_smaxData, 0);
	}*/

	return std::make_tuple(batchError, m_d_output);
}

float* SoftmaxLayer::backward(hipdnnHandle_t& handle, hipblasHandle_t& cublasHandle, float* d_loss_data, float* d_targets, float* d_onevec, float* previousLayerOutput) {
	// Softmax layer
	//SoftmaxLossBackprop <<< RoundUp(m_batchSize, m_batchSize), m_batchSize >>> (labels, m_output, m_batchSize, dloss_data);
	float *d_diffData, *d_gradData;
	Logger::instance()->writeLine("Softmax bwd");

	hipdnnTensorDescriptor_t diffTensorDesc;
	CheckError(hipdnnCreateTensorDescriptor(&diffTensorDesc), __FILE__, __LINE__);
	int dimA[3];
	int strideA[3];
	dimA[0] = m_batchSize;
	dimA[1] = m_inputDim;
	dimA[2] = 1;

	strideA[0] = dimA[2] * dimA[1];
	strideA[1] = dimA[2];
	strideA[2] = 1;
	CheckError(hipdnnSetTensorNdDescriptor(diffTensorDesc, HIPDNN_DATA_FLOAT, 3, dimA, strideA), __FILE__, __LINE__);

	/*float *loss_data = new float [m_batchSize*m_inputDim];
	//CheckError(hipMalloc((void**)&loss_data, m_batchSize*m_inputDim * sizeof(float)), __FILE__, __LINE__);
	CheckError(hipMemcpy(loss_data, dloss_data, m_batchSize*m_inputDim * sizeof(float), hipMemcpyDeviceToHost), __FILE__, __LINE__);*/
	//CheckError(hipDeviceSynchronize(), __FILE__, __LINE__);

	/*printDeviceVectorToFile(m_batchSize*m_inputDim, dloss_data, 0);
	printVectorToFile(m_batchSize*m_inputDim, loss_data, 0);*/

	dim3 b = { (unsigned int)RoundUp(m_batchSize,128), (unsigned int)m_inputDim, 1};
	dim3 t = { 128 , 1, 1 };

	CheckError(hipMalloc((void**)&d_diffData, m_batchSize*m_inputDim * sizeof(float)), __FILE__, __LINE__);

	SoftmaxLossKernel<<<b, t >>>(d_targets, d_loss_data, m_batchSize*m_inputDim);
	/*float* diffData = new float[m_batchSize*m_inputDim];
	for (int b = 0; b < m_batchSize; b++) {
		float* target = targets[b];
		for (int i = 0; i < m_inputDim; i++) {
			if(target[i] == 1.f)
				loss_data[b * m_inputDim + i] -= 1.f;
		}
	}*/

	//CheckError(hipMemcpy(dloss_data, loss_data, m_batchSize*m_inputDim * sizeof(float), hipMemcpyHostToDevice), __FILE__, __LINE__);
	//CheckError(hipDeviceSynchronize(), __FILE__, __LINE__);

	float scalVal = 1.0f / static_cast<float>(m_batchSize);
	CheckError(hipblasSscal(cublasHandle, m_inputDim * m_batchSize, &scalVal, d_loss_data, 1), __FILE__, __LINE__);
	
	/*Logger::instance()->writeLine("Output");
	printDeviceVectorToFile(2, m_output, 0);*/
	
	CheckError(hipMalloc((void**)&d_gradData, m_batchSize*m_inputDim * sizeof(float)), __FILE__, __LINE__);
	//CheckError(hipMemcpy(d_diffData, diffData, m_batchSize*m_inputDim * sizeof(float), hipMemcpyHostToDevice), __FILE__, __LINE__);
	
	
	CheckError(hipDeviceSynchronize(), __FILE__, __LINE__);
	
	float alpha(1), beta(0);

	CheckError(hipdnnSoftmaxBackward(handle,
		HIPDNN_SOFTMAX_ACCURATE,
		HIPDNN_SOFTMAX_MODE_CHANNEL,
		&alpha,
		diffTensorDesc,
		d_loss_data,
		diffTensorDesc,
		d_diffData,
		&beta,
		m_outputDesc,
		d_gradData), __FILE__, __LINE__);

	Logger::instance()->writeLine("d_loss_data");
	printDeviceVectorToFile(10, d_loss_data, 0);

	Logger::instance()->writeLine("d_diffData");
	printDeviceVectorToFile(10, d_diffData, 0);

	/*Logger::instance()->writeLine("\tdloss_data ======== 1");
	printDeviceVectorToFile(10, d_loss_data, 0);*/

	/*delete[] dimA;
	delete[] strideA;*/

	return d_loss_data;
}

void SoftmaxLayer::initWeights(hipdnnHandle_t &)
{

}

void SoftmaxLayer::initEpoch(hipdnnHandle_t & handle)
{
}

void SoftmaxLayer::updateWeight(hipblasHandle_t&, float)
{
}
