#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <cmath>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <random>
#include <thread>
#include <sstream>
#include <vector>
#include <tuple>
#include <>

#include "Layer.h"
#include "Network.h"
#include "Logger.h"
#include "GenericFunctions.h"

__global__ void PrepareDataKernel(float* d_batchData, float* d_batchTarget, int start, int end, float* d_input, float* d_targets) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx + start >= end) return;

	d_batchData[idx] = d_input[idx + start];
	d_batchTarget[2 * idx] = d_targets[2 * (idx + start)];
	d_batchTarget[(2 * (idx + 1)) - 1] = d_targets[(2 * (idx + start + 1)) - 1];
}

Network::Network() : Network(128, 0.01f, 4, 1, 1) {}

Network::Network(int batchSize, float learningRate, int inputSize, int outputDim, int seqLength = 50) : m_batchSize(batchSize), m_learningRate(learningRate), m_inputDim(inputSize), m_seqLength(seqLength), m_outputDim(outputDim){

	//
	// Listing GPU Devices
	//
	std::stringstream toWrite(std::stringstream::in | std::stringstream::out);
	int gpuNumbers;
	CheckError(hipGetDeviceCount(&gpuNumbers), __FILE__, __LINE__);
	int i = 0;
	hipDeviceProp_t prop;
	toWrite << "Cuda capable devices " << gpuNumbers << ":" << std::endl;
	for (i = 0; i < gpuNumbers; i++) {
		CheckError(hipGetDeviceProperties(&prop, i), __FILE__, __LINE__);
		toWrite << "\tdevice " << i << " (" << prop.name << ") : Proc " << prop.multiProcessorCount << ", Capabilities " << prop.major << "." << prop.minor << ", SmClock " << (float)prop.clockRate*1e-3 << " Mhz" << ", MemSize(Mb) " << (int)(prop.totalGlobalMem / (1024 * 1024)) << ", MemClock " << (float)prop.memoryClockRate*1e-3 << " Mhz" << std::endl;
	}
	m_gpuid = 0;

	//
	// Setting CUDA device
	//
	std::cout << "Using device " << m_gpuid << std::endl;
	toWrite << "Using device " << m_gpuid << std::endl;

	CheckError(hipSetDevice(m_gpuid), __FILE__, __LINE__);

	//
	// Getting CudNN version
	//

	size_t version = hipdnnGetVersion();
	std::cout << "CudNN version " << version << std::endl;
	toWrite << "CudNN version " << version << std::endl;

	//
	// Create CuDNN Handler
	//
	CheckError(hipdnnCreate(&m_handle), __FILE__, __LINE__);

	//
	// Create Cublas Handler
	//
	CheckError(hipblasCreate(&m_cublasHandle), __FILE__, __LINE__);
	Logger::instance()->writeLine(toWrite.str());
}

Network::~Network() {
	CheckError(hipdnnDestroy(m_handle), __FILE__, __LINE__);
	CheckError(hipblasDestroy(m_cublasHandle), __FILE__, __LINE__);
	CheckError(hipSetDevice(m_gpuid), __FILE__, __LINE__);
	CheckError(hipDeviceReset(), __FILE__, __LINE__);
}

void Network::train(float* d_data, float* d_labels, int epochNumber, int nbData)
{
	std::stringstream toWrite(std::stringstream::in | std::stringstream::out);
	toWrite << "==========================================================" << std::endl;
	toWrite << "=========================Training=========================" << std::endl;
	toWrite << "==========================================================" << std::endl;
	Logger::instance()->writeLine(toWrite.str());

	std::cout << "Training..." << std::endl;

	float timeTraining;
	hipEvent_t start, stop;
	CheckError(hipEventCreate(&start), __FILE__, __LINE__);
	CheckError(hipEventCreate(&stop), __FILE__, __LINE__);
	CheckError(hipEventRecord(start), __FILE__, __LINE__);

	//
	//  Initialize weights of each layer
	//
	for (Layer& l : m_layers)
		l.initWeights(m_handle);

	CheckError(hipDeviceSynchronize(), __FILE__, __LINE__);

	int nbBatch = std::ceil(double(nbData) / double(m_batchSize));
	toWrite.str("");
	toWrite.clear();
	toWrite << "Number of iteration per epoch : " << nbBatch << std::endl;
	Logger::instance()->writeLine(toWrite.str());
	std::cout << nbBatch << " batchs to run per iteration" << std::endl;

	for (int iter = 0; iter < epochNumber; iter++)
	{
		trainEpoch(iter, epochNumber, nbBatch, nbData, d_data, d_labels);
	}

	CheckError(hipDeviceSynchronize(), __FILE__, __LINE__);
	CheckError(hipEventRecord(stop), __FILE__, __LINE__);
	CheckError(hipEventSynchronize(stop), __FILE__, __LINE__);
	CheckError(hipEventElapsedTime(&timeTraining, start, stop), __FILE__, __LINE__);

	toWrite.str("");
	toWrite.clear();
	toWrite << "==========================================================" << std::endl;
	toWrite << "=======================End Training=======================" << std::endl;
	toWrite << "==========================================================" << std::endl;
	toWrite << "Time " << timeTraining << "ms";
	Logger::instance()->writeLine(toWrite.str());
	printf("Training time (%3.0f ms)\n", timeTraining);
}

void Network::addLayer(Layer& l)
{
	m_layers.push_back(l);
}

hipdnnHandle_t & Network::getHandle()
{
	return m_handle;
}

void Network::trainEpoch(int epoch, int nbEpoch, int nbBatch, int nbData, float* d_input, float* d_targets)
{
	std::stringstream toWrite(std::stringstream::in | std::stringstream::out);
	std::cout << "Epoch " << (epoch + 1) << " / " << nbEpoch << std::endl;
	toWrite << "==========================================================" << std::endl;
	Logger::instance()->writeLine(toWrite.str());
	int curNbBatch = 0;
	float error = 0.f;

	//
	//  Initialize weights of each layer
	//
	for (Layer& l : m_layers)
		l.initEpoch(m_handle);

	for (int b = 0; b < nbBatch; b++) {
		float *d_bData, *d_bTargets, *d_onevec;
		/*bData = new float[m_inputDim * m_batchSize];
		bTargets = new float*[m_inputDim * m_batchSize];*/
		CheckError(hipMalloc((void**)&d_bData, m_seqLength * m_inputDim * m_batchSize * sizeof(float)), __FILE__, __LINE__);
		CheckError(hipMalloc((void**)&d_bTargets, 2 * m_seqLength * m_inputDim * m_batchSize * sizeof(float)), __FILE__, __LINE__);

		// Get data and target for current batch
		prepareData(d_input, d_targets, b, d_bData, d_bTargets);

		curNbBatch += m_batchSize;
		toWrite.str("");
		toWrite.clear();
		toWrite << "Epoch " << (epoch + 1) << " / " << nbEpoch << " [" << curNbBatch << " / " << nbData << "]";
		Logger::instance()->writeLine(toWrite.str());

		CheckError(hipMalloc(&d_onevec, sizeof(float)* m_batchSize), __FILE__, __LINE__);
		FillVec(m_batchSize, d_onevec, 0.f);
		// Forward pass + get error
		auto resultFwd = forward(d_bTargets, d_bTargets, d_onevec);

		//Backward pass
		backward(std::get<1>(resultFwd), d_bTargets, d_onevec);

		// Updating weights
		for (auto it = m_layers.begin(); it != m_layers.end(); ++it) {
			it->get().updateWeight(m_cublasHandle, m_learningRate);
		}

		toWrite.str("");
		toWrite.clear();
		toWrite << "\tError " << std::get<0>(resultFwd);
		Logger::instance()->writeLine(toWrite.str());

		error += std::get<0>(resultFwd);
		//validateBatch();

		CheckError(hipFree(d_bData), __FILE__, __LINE__);
		CheckError(hipFree(d_bTargets), __FILE__, __LINE__);
		CheckError(hipFree(d_onevec), __FILE__, __LINE__);
	}
	error /= nbBatch;
	toWrite.str("");
	toWrite.clear();
	toWrite << "Batch Error " << error;
	Logger::instance()->writeLine(toWrite.str());
}

std::tuple<float, std::vector<float*>> Network::forward(float* d_input, float* d_target, float* d_onevec)
{
	float error = 0;
	std::vector<std::reference_wrapper<Layer>>::iterator it = m_layers.begin();
	std::vector<float*> output;
	std::tuple<float, float*> result = it->get().forward(m_handle, m_cublasHandle, d_input, d_target, d_onevec);
	output.push_back(std::get<1>(result));
	it++;
	for (; it != m_layers.end(); ++it) {
		result = it->get().forward(m_handle, m_cublasHandle, output.back(), d_target, d_onevec);
		output.push_back(std::get<1>(result));
		error += std::get<0>(result);
	}
	return std::tuple<float, std::vector<float*>>(error, output);
}

void Network::backward(std::vector<float*> fwdOutput, float* target, float* d_onevec)
{
	float* dloss_data;

	CheckError(hipMalloc(&dloss_data, sizeof(float) * m_batchSize * m_outputDim), __FILE__, __LINE__);
	CheckError(hipMemcpyAsync(dloss_data, fwdOutput.back(), sizeof(float) * m_batchSize * m_outputDim, hipMemcpyDeviceToDevice), __FILE__, __LINE__);
	
	// Accounting for batch size in SGD
	//CheckError(hipblasSscal(m_cublasHandle, m_outputDim * m_batchSize, &scalVal, dloss_data, 1), __FILE__, __LINE__);
	std::vector<std::reference_wrapper<Layer>>::reverse_iterator it = m_layers.rbegin();
	float* result = it->get().backward(m_handle, m_cublasHandle, dloss_data, target, d_onevec, fwdOutput[fwdOutput.size() - 1]);

	++it;

	for (int i = fwdOutput.size() - 2; it != m_layers.rend(); ++it, i--) {
		result = it->get().backward(m_handle, m_cublasHandle, result, target, d_onevec, fwdOutput[i]);
	}
}

void Network::prepareData(float* d_input, float* d_target, int b, float* d_batchData, float* d_batchTarget) {

	int start = b*m_inputDim;
	int end = (b + m_batchSize)*m_inputDim;
	int size = end - start;

	PrepareDataKernel <<< RoundUp(size, 128), 128 >>> (d_batchData, d_batchTarget, start, end, d_input, d_target);

	//
	//	Loading data for this batch
	//
	/*int srcSize = m_inputDim * m_batchSize;
	float* batchData = new float[srcSize];

	for (int i = b*m_inputDim, j = 0; i < (b + m_batchSize)*m_inputDim; i++, j++) {
		batchData[j] = input[i];
		bTarget[j] = target[i];
	}
	hipMemcpy(d_batchData, batchData, m_batchSize*m_inputDim * sizeof(float), hipMemcpyHostToDevice);
	delete[] batchData;*/
}